#include "hip/hip_runtime.h"
#include "rmse.h"
// #define M 6
// #define N 5
// #define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))

//Cui will look like this:
//  Cui[0] = indptr - pointer into indices/data showing where each row starts
//  Cui[1] = indices - which columns in that row exist
//  Cui[2] = data - what's in that column

double rmse(hipblasHandle_t handle, double* user_factors, double* item_factors,
            int* rows, int* cols, double* ratings, int num_things, int factors) {
    double error = 0;
    hipblasStatus_t stat;
    hipError_t err;

    for (int k = 0; k < num_things; ++k) {
        printf("rmse iter %d\n", k);
        hipDeviceSynchronize();
        int uid = rows[k];
        int iid = cols[k];
        double rating = ratings[k];
        printf("%d\n", uid);
        printf("%d\n", iid);
        printf("%f\n", rating);

        double* user;
        double user_host[factors];
        err = hipMallocManaged(&user, factors*sizeof(double));
        if (err != hipSuccess) {
          printf("%s\n", hipGetErrorString(err));
          hipFree(user);
          return -1;
        }
        hipMemcpy(user, &user_factors[uid*factors], factors*sizeof(double), hipMemcpyDeviceToDevice);
        hipMemcpy(user_host, &user_factors[uid*factors], factors*sizeof(double), hipMemcpyDeviceToHost);
        double* item;
        double item_host[factors];
        err = hipMallocManaged(&item, factors*sizeof(double));
        if (err != hipSuccess) {
          printf("%s\n", hipGetErrorString(err));
          hipFree(user);
          hipFree(item);
          return -1;
        }
        hipMemcpy(item, &item_factors[iid*factors], factors*sizeof(double), hipMemcpyDeviceToDevice);
        hipMemcpy(item_host, &item_factors[iid*factors], factors*sizeof(double), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        for (int i = 0; i < factors; ++i) {
          printf("%f\n", user_host[i]);
          printf("%f\n", item_host[i]);
        }
        // printf("%f\n", user[0]);
        // printf("%f\n", item[0]);

        double guess;

        stat = hipblasDdot(handle, factors, user, 1, item, 1, &guess);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            printf ("ddot failed\n");
            printf("%d\n", stat);
            return -1;
        }
        error += std::pow(abs(rating-guess), 2);
        printf("rating %f\n", rating);
        printf("guess %f\n", guess);
        printf("error %f\n", error);
        hipFree(user);
        hipFree(item);
    }
    return std::sqrt(error/num_things);
}
