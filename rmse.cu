#include "hip/hip_runtime.h"
#include "rmse.h"
// #define M 6
// #define N 5
// #define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))

//Cui will look like this:
//  Cui[0] = indptr - pointer into indices/data showing where each row starts
//  Cui[1] = indices - which columns in that row exist
//  Cui[2] = data - what's in that column

double rmse(hipblasHandle_t handle, double* user_factors, double* item_factors,
            int* rows, int* cols, double* ratings, int num_things, int factors) {
    double error = 0;
    hipblasStatus_t stat;

    for (int k = 0; k < num_things; ++k) {
        printf("rmse iter %d\n", k);
        hipDeviceSynchronize();
        int uid = rows[k];
        int iid = cols[k];
        double rating = ratings[k];
        printf("%d\n", uid);
        printf("%d\n", iid);
        printf("%f\n", rating);

        double* user = &user_factors[uid*factors];
        double* item = &item_factors[iid*factors];
        hipDeviceSynchronize();

        // printf("%f\n", user[0]);
        // printf("%f\n", item[0]);

        double guess;

        stat = hipblasDdot(handle, factors, user, 1, item, 1, &guess);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            printf ("ddot failed\n");
            printf("%d\n", stat);
            return -1;
        }
        error += std::pow((rating-guess), 2);
        printf("error %f\n", error);
    }
    return std::sqrt(error/num_things);
}
