#include "hip/hip_runtime.h"
#ifndef RECOMMEND_CU
#define RECOMMEND_CU

// #include "recommend.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "hipsparse.h"
#include <cmath>
#include <vector>
#include <fstream>
#include <iomanip>
#include <cstring>
// #define M 6
// #define N 5
// #define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1)) 

//Cui will look like this:
//  Cui[0] = indptr - pointer into indices/data showing where each row starts
//  Cui[1] = indices - which columns in that row exist
//  Cui[2] = data - what's in that column

double rmse(double* user_factors, double* item_factors, int* rows, int* cols, double* ratings, int num_things, int factors) {
    double error = 0;
    hipblasHandle_t handle;
    hipblasStatus_t err;

    for (int k = 0; k < num_things; ++k) {
        int uid = rows[k];
        int iid = cols[k];
        double rating = ratings[k];

        double* user = user_factors[uid*k];
        double* item = item_factors[iid*k];

        double guess;

        err = hipblasDdot(handle, factors, user, 1, item, 1, &guess);
        error += std::pow((rating-guess), 2);
    }
    return std::sqrt(error/num_things);
}

#endif