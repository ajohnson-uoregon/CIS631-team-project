#include "hip/hip_runtime.h"
#include "recommend.h"
#include<iostream>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "hipsparse.h"
// #define M 6
// #define N 5
// #define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1)) 

std::list<int> recommend(int userid, std::vector<std::vector<int>> user_items, std::vector<std::vector< int>> user_factors, std::vector<std::vector< int>> item_factors, int N)
{
    int users = user_items.size();
    int items = user_items[0].size();
    int factors = user_items[0].size();
    std::cout << users << std::endl;
    std::list<int> ans = {3};
    return ans;
}

double calculate_loss(double** Cui, double** X, double** Y, double reg) {
    int users = X.size();
    int items = Y.size();
    int factors = X[0].size();

    int loss = 0;
    int total_confidence = 0;
    int item_norm = 0;
    int user_norm = 0;
    hipblasStatus_t err;

    // do transpose
    err = cublas_Dgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, items, factors,
              1, Y, 0, 0, Y, 0, YtY, 0);

    for (int u = 0; u < users; ++u) {
        double temp = 1.0;

        double* r;
        double* Xu = X[u];

        err = hipblasDgemv(handle, HIPBLAS_OP_N, items, factors,
                  1, Y, 0, Xu, 1, 0, r, 1);

        //TODO: how to sparse
        cols, vals = Cui[u];
        for (index = 0; index < cols.size(); ++index) {
            int i = cols[index];
            double confidence = vals[index];

            double* Yi = Y[i];

            double d;
            err = hipblasDdot(handle, factors, Yi, 1, Xu, 1, d);

            temp = (confidence - 1)*d - (2*confidence);
            
            err = hipblasDaxpy(handle, factors, temp, Yi, 1, r, 1);

            total_confidence += confidence;
            loss += confidence;
        }

        double other_temp;

        err = hipblasDdot(handle, factors, r, 1, Xu, 1, other_temp);
        loss += other_temp;
        
        err = hipblasDdot(handle, factors, Xu, 1, Xu, 1, other_temp);
        user_norm += other_temp;
    }

    for (int i = 0; i < items; ++i) {
        double* Yi = Y[i];

        double other_temp;
 
        err = hipblasDdot(handle, factors, Yi, 1, Yi, 1, other_temp);
        item_norm += other_temp;
    }

    loss += reg * (item_norm + user_norm);

    //TODO: how nnz
    nnz = Cui.nnz;

    return loss / (total_confidence + users * items - nnz);
}
/*
def rmse(values, user_factors, item_factors):
    error = 0
    indptr, indices, data = values
    num_things = 0
    for uid in range(len(indptr) - 2):
        row_start = indptr[uid]
        row_end = indptr[uid+1]
        for index in range(row_start,row_end):
            iid = indices[index]
            rating = data[index]

            user = petsc.Vec()
            user.createSeq(factors)
            user.setValues(list(range(factors)), user_factors.getValues([uid], list(range(factors))))
            user.assemble()

            item = petsc.Vec()
            item.createSeq(factors)
            item.setValues(list(range(factors)), item_factors.getValues([iid], list(range(factors))))
            item.assemble()

            guess = user.dot(item)

            error += (rating-guess)**2
            num_things += 1

    return math.sqrt(error/num_things)
*/
