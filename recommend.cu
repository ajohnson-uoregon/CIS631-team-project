#include "hip/hip_runtime.h"
#include "recommend.h"
#include<iostream>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "hipsparse.h"
#include <cmath>
#include <vector>
#include <fstream>
#include <iomanip>
// #define M 6
// #define N 5
// #define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1)) 
/*
std::list<int> recommend(int userid, std::vector<std::vector<int> > user_items, std::vector<std::vector<int> > user_factors, std::vector<std::vector<int> > item_factors, int N)
{
    int users = user_items.size();
    int items = user_items[0].size();
    int factors = user_items[0].size();
    std::cout << users << std::endl;
    std::list<int> ans = {3};
    return ans;
}
*/
//Cui will look like this:
//  Cui[0] = indptr - pointer into indices/data showing where each row starts
//  Cui[1] = indices - which columns in that row exist
//  Cui[2] = data - what's in that column

int main(int argc, char** argv) {
    char* fname = argv[1];
    int iterations = strtol(argv[2], NULL, 10);
    int factors = strtol(argv[3], NULL, 10);

    int users = 0;
    int items = 0;

    printf("%s\n", fname);

    FILE* fp;
    char* line = NULL;
    ssize_t read;
    size_t len = 0;
    std::vector<int> indptr;
    std::vector<int> indices;
    std::vector<double> data;

    fp = fopen(fname, "r");
    printf("done with setup\n");

    while ((read = getline(&line, &len, fp)) != -1) {
        int l = strlen(line);
        if (line[l-2] == ':') {
            indptr.push_back(indices.size());
        }
        else {
            int col = strtol(strtok(line, ","), NULL, 10);
            double rating = strtol(strtok(NULL, ","), NULL, 10);

            indices.push_back(col);
            data.push_back(rating);

            if (col > users) {
                users = col;
            }
        }
    }
    fclose(fp);

    items = indptr.size();
    users += 1;

    printf("users: %d\n", users);
    printf("items: %d\n", items);
    printf("factors: %d\n", factors);

    return 0;

}

double calculate_loss(double** Cui, double** X, double** Y, double reg, 
  int users, int items, int factors, int nnz) {

    int loss = 0;
    int total_confidence = 0;
    int item_norm = 0;
    int user_norm = 0;
    hipblasStatus_t err;
    hipblasHandle_t handle;

    double** YtY;
    const double alpha = 1;
    const double beta = 0;

    // do transpose
    err = hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, items, factors,
              &alpha, Y[0], items, &beta, Y[0], items, YtY[0], factors);
    
    for (int u = 0; u < users; ++u) {
        double temp = 1.0;

        double* r;
        double* Xu = X[u];
        

        err = hipblasDgemv(handle, HIPBLAS_OP_N, items, factors,
                  &alpha, Y[0], items, Xu, 1, &beta, r, 1);
        
        int rowStart = Cui[0][u];
        int rowEnd = Cui[0][u+1];

        int cols[rowEnd-rowStart];
        memcpy(cols, &Cui[1][rowStart], rowEnd-rowStart);
        //int* cols = Cui[1][rowStart:rowEnd];
        
        double vals[rowEnd-rowStart];
        memcpy(vals, &Cui[2][rowStart], rowEnd-rowStart);
        //double* vals = Cui[2][rowStart:rowEnd];
        for (int index = 0; index < rowEnd-rowStart; ++index) {
            int i = cols[index];
            double confidence = vals[index];

            double* Yi = Y[i];

            double d;
            err = hipblasDdot(handle, factors, Yi, 1, Xu, 1, &d);
            temp = (confidence - 1)*d - (2*confidence);
            
            err = hipblasDaxpy(handle, factors, &temp, Yi, 1, r, 1);
            total_confidence += confidence;
            loss += confidence;
        }

        double other_temp;

        err = hipblasDdot(handle, factors, r, 1, Xu, 1, &other_temp);
        loss += other_temp;
        
        err = hipblasDdot(handle, factors, Xu, 1, Xu, 1, &other_temp);
        user_norm += other_temp;
    }

    for (int i = 0; i < items; ++i) {
        double* Yi = Y[i];

        double other_temp;
 
        err = hipblasDdot(handle, factors, Yi, 1, Yi, 1, &other_temp);
        item_norm += other_temp;
    }

    loss += reg * (item_norm + user_norm);

    return loss / (total_confidence + users * items - nnz);
}

double rmse(double** user_factors, double** item_factors, int* rows, int* cols, double* ratings, int num_things, int factors) {
    double error = 0;
    hipblasHandle_t handle;
    hipblasStatus_t err;

    for (int k = 0; k < num_things; ++k) {
        int uid = rows[k];
        int iid = cols[k];
        double rating = ratings[k];

        double* user = user_factors[uid];
        double* item = item_factors[iid];

        double guess;

        err = hipblasDdot(handle, factors, user, 1, item, 1, &guess);
        error += std::pow((rating-guess), 2);
    }
    return std::sqrt(error/num_things);
}
