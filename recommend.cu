#include "hip/hip_runtime.h"
#include "recommend.h"
#include<iostream>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "hipsparse.h"
// #define M 6
// #define N 5
// #define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1)) 
/*
std::list<int> recommend(int userid, std::vector<std::vector<int> > user_items, std::vector<std::vector<int> > user_factors, std::vector<std::vector<int> > item_factors, int N)
{
    int users = user_items.size();
    int items = user_items[0].size();
    int factors = user_items[0].size();
    std::cout << users << std::endl;
    std::list<int> ans = {3};
    return ans;
}
*/
//Cui will look like this:
//  Cui[0] = indptr - pointer into indices/data showing where each row starts
//  Cui[1] = indices - which columns in that row exist
//  Cui[2] = data - what's in that column

double calculate_loss(double** Cui, double** X, double** Y, double reg, 
  int users, int items, int factors, int nnz) {

    int loss = 0;
    int total_confidence = 0;
    int item_norm = 0;
    int user_norm = 0;
    hipblasStatus_t err;
    hipblasHandle_t handle;

    double** YtY;
    const double alpha = 1;
    const double beta = 0;

    // do transpose
    err = hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, items, factors,
              &alpha, Y[0], items, &beta, Y[0], items, YtY[0], factors);
    
    for (int u = 0; u < users; ++u) {
        double temp = 1.0;

        double* r;
        double* Xu = X[u];
        

        err = hipblasDgemv(handle, HIPBLAS_OP_N, items, factors,
                  &alpha, Y[0], items, Xu, 1, &beta, r, 1);
        
        int rowStart = Cui[0][u];
        int rowEnd = Cui[0][u+1];

        int cols[rowEnd-rowStart];
        memcpy(cols, &Cui[1][rowStart], rowEnd-rowStart);
        //int* cols = Cui[1][rowStart:rowEnd];
        
        double vals[rowEnd-rowStart];
        memcpy(vals, &Cui[2][rowStart], rowEnd-rowStart);
        //double* vals = Cui[2][rowStart:rowEnd];
        for (int index = 0; index < rowEnd-rowStart; ++index) {
            int i = cols[index];
            double confidence = vals[index];

            double* Yi = Y[i];

            double d;
            err = hipblasDdot(handle, factors, Yi, 1, Xu, 1, &d);
            temp = (confidence - 1)*d - (2*confidence);
            
            err = hipblasDaxpy(handle, factors, &temp, Yi, 1, r, 1);
            total_confidence += confidence;
            loss += confidence;
        }

        double other_temp;

        err = hipblasDdot(handle, factors, r, 1, Xu, 1, &other_temp);
        loss += other_temp;
        
        err = hipblasDdot(handle, factors, Xu, 1, Xu, 1, &other_temp);
        user_norm += other_temp;
    }

    for (int i = 0; i < items; ++i) {
        double* Yi = Y[i];

        double other_temp;
 
        err = hipblasDdot(handle, factors, Yi, 1, Yi, 1, &other_temp);
        item_norm += other_temp;
    }

    loss += reg * (item_norm + user_norm);

    return loss / (total_confidence + users * items - nnz);
}
/*
def rmse(values, user_factors, item_factors):
    error = 0
    indptr, indices, data = values
    num_things = 0
    for uid in range(len(indptr) - 2):
        row_start = indptr[uid]
        row_end = indptr[uid+1]
        for index in range(row_start,row_end):
            iid = indices[index]
            rating = data[index]

            user = petsc.Vec()
            user.createSeq(factors)
            user.setValues(list(range(factors)), user_factors.getValues([uid], list(range(factors))))
            user.assemble()

            item = petsc.Vec()
            item.createSeq(factors)
            item.setValues(list(range(factors)), item_factors.getValues([iid], list(range(factors))))
            item.assemble()

            guess = user.dot(item)

            error += (rating-guess)**2
            num_things += 1

    return math.sqrt(error/num_things)
*/
