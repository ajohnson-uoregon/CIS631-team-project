#include<iostream> 
#include "recommend.h"

using namespace std;
 
void print(const list<int>& s) {
	list<int>::const_iterator i;
	for( i = s.begin(); i != s.end(); ++i)
		cout << *i << " ";
	cout << endl;
}

int main()
{
    vector<vector <int>> t1({{1,2,3}});
    cout<<"Hello World \n";
    const list<int> ans = (recommend(1, t1, t1, t1, 1));
    print(ans);
    return 0;
}