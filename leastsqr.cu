#include "hip/hip_runtime.h"
#include "leastsqr.h"

void GPU_fill_rand(double *A, const int nr_rows_A, const int nr_cols_A)
{
    hiprandGenerator_t prng;
    hiprandStatus_t stat;

    stat = hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
    if (stat != HIPRAND_STATUS_SUCCESS) {
        printf("error at %s:%d\n",__FILE__, __LINE__);
        return;
    }
    stat = hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());
    if (stat != HIPRAND_STATUS_SUCCESS) {
        printf("error at %s:%d\n",__FILE__, __LINE__);
        return;
    }

    stat = hiprandGenerateUniformDouble(prng, A, nr_rows_A * nr_cols_A);
    if (stat != HIPRAND_STATUS_SUCCESS) {
        printf("error at %s:%d\n",__FILE__, __LINE__);
        return;
    }

}
// borrowed from https://stackoverflow.com/questions/21112725/cuda-fill-an-matrix-with-random-values-between-a-and-b
int iDivUp(int a, int b) { return ((a % b) != 0) ? (a / b + 1) : (a / b); }

__global__ void shiftDiagonal(double *A, double reg, int num_cols) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < num_cols) {
      A[num_cols*tid + tid] += reg;
    }
}


void least_squares (hipblasHandle_t handle, int* indptr, int* indic, double* data, int users,
                        int items, int factors,
                        double* x, double* y,
                        double reg, int istart, int iend)
{
    hipError_t err;
    hipblasStatus_t stat;
    hipsolverHandle_t solve_handle;
    const double alpha = 1;
    const double beta = 0;

    //double yt[factors][items];

    double* yt;

    err = hipMallocManaged(&yt, factors*items*sizeof(double));
    if (err != hipSuccess) {
      printf("%s\n", hipGetErrorString(err));
      hipFree(yt);
      return;
    }

    stat = hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, factors, items,
        &alpha, y, items, &beta, y, items, yt, factors);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("dgeam failed\n");
        hipFree(yt);
        return;
    }

    hipblasSideMode_t mode = HIPBLAS_SIDE_LEFT;

    for (int i =0; i < users; ++i) {
        printf("starting for loop\n");
        printf("users %d i %d\n", users, i);
        hipDeviceSynchronize();

        int rowStart = indptr[i];
        int rowEnd = indptr[i+1];;

        printf("rowstart %d\n", rowStart);
        printf("rowEnd %d\n", rowEnd);

        printf("merp\n");
        int cols[rowEnd-rowStart];
        printf("%d\n", indic[rowStart]);

        memcpy(cols, &indic[rowStart], (rowEnd-rowStart)*sizeof(int));
        // err = hipMemcpy(cols, ptrptr, rowEnd-rowStart, hipMemcpyDeviceToHost);
        // if (err != hipSuccess) {
        //   printf("first copy %s\n", hipGetErrorString(err));
        //   return;
        // }
        printf("%d\n", cols[0]);
        printf("done w first memcpy\n");
        double vals[rowEnd-rowStart];
        memcpy(vals, &data[rowStart], (rowEnd-rowStart)*sizeof(double));
        // err = hipMemcpy(vals, &data[rowStart], rowEnd-rowStart, hipMemcpyDeviceToHost);
        // if (err != hipSuccess) {
        //   printf("first copy %s\n", hipGetErrorString(err));
        //   return;
        // }
        printf("done with memcpys\n");

        double diagVec[items];
        std::fill_n(diagVec, items, 0);
        //hipDeviceSynchronize();
        for(int iter = 0; iter < (rowEnd-rowStart); ++iter)
        {
            printf("loooop\n");
            printf("%d\n", cols[iter]);
            printf("%f\n", vals[iter]);
            diagVec[cols[iter]] = vals[iter];
        }
        printf("making device vec\n");
        double* diagVec_dev;

        err = hipMallocManaged(&diagVec_dev, items*sizeof(double));
        if (err != hipSuccess) {
          printf("%s\n", hipGetErrorString(err));
          hipFree(yt);
          hipFree(diagVec_dev);
          return;
        }
        err = hipMemcpy(diagVec_dev, diagVec, items, hipMemcpyHostToDevice);
        if (err != hipSuccess) {
          printf("%s\n", hipGetErrorString(err));
          hipFree(yt);
          hipFree(diagVec_dev);
          return;
        }
        printf("making ansC\n");

        double* ansC; //[items][factors];

        err = hipMallocManaged(&ansC, items*factors*sizeof(double));
        if (err != hipSuccess) {
          printf("%s\n", hipGetErrorString(err));
          hipFree(yt);
          hipFree(diagVec_dev);
          hipFree(ansC);
          return;
        }

        printf("dgmm\n");
        stat = hipblasDdgmm(handle, mode, items, factors, y, items, diagVec, 1, ansC, items);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            printf ("ddgmm failed\n");
            hipFree(yt);
            hipFree(diagVec_dev);
            hipFree(ansC);
            return;
        }
        double* ytcy; //[factors][factors];

        err = hipMallocManaged(&ytcy, factors*factors*sizeof(double));
        if (err != hipSuccess) {
          printf("%s\n", hipGetErrorString(err));
          hipFree(yt);
          hipFree(diagVec_dev);
          hipFree(ansC);
          hipFree(ytcy);
          return;
        }

        printf("gemm\n");
        stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, factors, factors,
                        items, &alpha, yt, factors, ansC, items, &beta, ytcy, factors);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            printf ("dgemm failed\n");
            hipFree(yt);
            hipFree(diagVec_dev);
            hipFree(ansC);
            hipFree(ytcy);
            return;
        }
        hipDeviceSynchronize();
        printf("setting diagonal\n");
        // for (int k = 0; k < factors; ++k)
        // {
        //     printf("%d\n",k);
        //     printf("%f\n", ytcy[0]);
        //     ytcy[factors*k +k] += reg;
        // }
        dim3 threads(32);
        dim3 blocks(iDivUp(factors*factors, 32));
        shiftDiagonal<<<blocks, threads>>>(ytcy, reg, factors);
        printf("done w diagonal\n");
        hipDeviceSynchronize();

        double* ytcu; //[factors];
        printf("kjdfjkfdjkfd\n");
        err = hipMallocManaged(&ytcu, factors*sizeof(double));
        if (err != hipSuccess) {
          printf("%s\n", hipGetErrorString(err));
          hipFree(yt);
          hipFree(diagVec_dev);
          hipFree(ansC);
          hipFree(ytcy);
          hipFree(ytcu);
          return;
        }
        printf("gemv\n");
        stat = hipblasDgemv (handle, HIPBLAS_OP_N, factors, items,
                        &alpha, yt, factors, diagVec_dev, 1, &beta, ytcu, 1);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            printf ("dgemm failed\n");
            hipFree(yt);
            hipFree(diagVec_dev);
            hipFree(ansC);
            hipFree(ytcy);
            hipFree(ytcu);
            return;
        }
        printf("solver fun times\n");
        int Lwork;
        hipsolverDnDgetrf_bufferSize(solve_handle, factors, factors, ytcy, factors, &Lwork);
        int* rfOut;
        double workspace[Lwork];
        hipsolverDnDgetrf(solve_handle, factors, factors, ytcy, factors, workspace, NULL, rfOut);
        int* rsOut;
        hipsolverDnDgetrs(solve_handle, HIPBLAS_OP_N, factors, 1, ytcy, factors, NULL, ytcu, factors, rsOut);
        memcpy(&x[(i*factors)], ytcu, factors);
        //x[i] = ytcu;
        hipFree(diagVec_dev);
        hipFree(ansC);
        hipFree(ytcy);
        hipFree(ytcu);
    }
    printf("done\n");
    hipFree(yt);
}
