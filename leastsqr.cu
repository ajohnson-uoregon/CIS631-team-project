#include "hip/hip_runtime.h"
#ifndef LEASTSQR_CU
#define LEASTSQR_CU
#include <cstring>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <hipsolver.h>
#include <hiprand.h>

void GPU_fill_rand(double *A, int nr_rows_A, int nr_cols_A)
{
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);

    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

    hiprandGenerateUniformDouble(prng, A, nr_rows_A * nr_cols_A);
}


void least_squares (int* indptr, int* indic, double* data, int users, 
                        int items, int factors,
                        double** x, double** y, 
                        double reg, int istart, int iend) 
{
    hipblasStatus_t err;
    hipblasHandle_t handle;
    hipsolverHandle_t solve_handle;
    const double alpha = 1;
    const double beta = 0;
    double yt[factors][items];
    
    err = hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, items, factors,
        &alpha, y[0], items, &beta, y[0], items, yt[0], factors);
    
    hipblasSideMode_t mode = HIPBLAS_SIDE_LEFT;

    for (int i =0; i < users; ++i) {
        
        int rowStart = indptr[i];
        int rowEnd = indptr[i+1];

        int cols[rowEnd-rowStart];
        memcpy(cols, &indic[rowStart], rowEnd-rowStart);
        
        double vals[rowEnd-rowStart];
        memcpy(vals, &data[rowStart], rowEnd-rowStart);

        double diagVec[items];
        std::fill_n(diagVec, items, 0);

        for(int iter = 0; iter < (rowEnd-rowStart); ++iter) 
        {
            diagVec[cols[iter]] = vals[iter];
        }   
        double ansC[items][factors];

        hipblasDdgmm(handle, mode, items, factors, y[0], items, diagVec, 1, ansC[0], items);
        double ytcy[factors][factors];

        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, factors, factors, 
                        items, &alpha, yt[0], factors, ansC[0], items, &beta, ytcy[0], factors);
        for (int k = 0; k < factors; ++k) 
        {
            ytcy[k][k] += reg;
        }

        double ytcu[factors];
        hipblasDgemv (handle, HIPBLAS_OP_N, factors, items,
                        &alpha, yt[0], factors, diagVec, 1, &beta, ytcu, 1);
        int Lwork;
        hipsolverDnDgetrf_bufferSize(solve_handle, factors, factors, ytcy[0], factors, &Lwork);
        int* rfOut;
        double workspace[Lwork];
        hipsolverDnDgetrf(solve_handle, factors, factors, ytcy[0], factors, workspace, NULL, rfOut);
        int* rsOut;
        hipsolverDnDgetrs(solve_handle, HIPBLAS_OP_N, factors, 1, ytcy[0], factors, NULL, ytcu, factors, rsOut);
        x[i] = ytcu;
    }
}
#endif