#include <cstring>
#include <algoithm>
double** least_squares (int* indptr, int* indic, double* data, int users, 
                        int items, int factors,
                        double** x, double** y, 
                        double reg, int istart, int iend) 
{
    hipblasStatus_t err;
    hipblasHandle_t handle;
    const double alpha = 1;
    const double beta = 0;
    double[factors][items] yt;
    err = hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, items, factors,
        &alpha, y[0], items, &beta, y[0], items, yt[0], factors);
    
    hipblasSideMode_t mode = HIPBLAS_SIDE_LEFT;

    for (int i =0; i < users; ++i) {
        
        int rowStart = indptr[i];
        int rowEnd = intptr[i+1];

        int cols[rowEnd-rowStart];
        memcpy(cols, &indic[rowStart], rowEnd-rowStart);
        
        double vals[rowEnd-rowStart];
        memcpy(vals, &data[rowStart], rowEnd-rowStart);

        double[items] diagVec;
        fill(diagVec.start(), diagVec.end(), 0);

        for(int iter = 0; iter < (rowEnd-rowStart); ++iter) 
        {
            diagVec[cols[iter]] = vals[iter];
        }   
        double[items][factors] ansC;

        hipblasDdgmm(handle, mode, items, factors, y[0], items, diagVec, 1, ansC, items);
        double[factors][factors] ytcy;

        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, factors, factors, 
                        items, &alpha, yt, factors, ansC, items, &beta, ytcy, factors);
        for (int i = 0; i < factors; ++i) 
        {
            ytcy[i][i] += reg;
        }

        double[factors] ytcu;
        hipblasDgemv (handle, HIPBLAS_OP_N, HIPBLAS_OP_N, factors, items,
                        &alpha, yt, factors, diagVec, 1, &beta, ytcu, 1);
        int* Lwork;
        hipsolverDnDgetrf_bufferSize(handle, factors, factors, ytcy, facotrs, Lwork);
        int* rfOut;
        hipsolverDnDgetrf(handle, factors, factors, ytcy, factors, Lwork, NULL, rfOut);
        int* rsOut;
        hipsolverDnDgetrs(handle, HIPBLAS_OP_N, factors, 1, ytcy, factors, NULL, ytcu, factors, rsOut);
        x[i] = ytcu;
    }

    return x;

}