#include "hip/hip_runtime.h"
#include "leastsqr.h"

void GPU_fill_rand(double *A, const int nr_rows_A, const int nr_cols_A)
{
    hiprandGenerator_t prng;
    hiprandStatus_t stat;

    stat = hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
    if (stat != HIPRAND_STATUS_SUCCESS) {
        printf("error at %s:%d\n",__FILE__, __LINE__);
        return;
    }
    stat = hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());
    if (stat != HIPRAND_STATUS_SUCCESS) {
        printf("error at %s:%d\n",__FILE__, __LINE__);
        return;
    }

    stat = hiprandGenerateUniformDouble(prng, A, nr_rows_A * nr_cols_A);
    if (stat != HIPRAND_STATUS_SUCCESS) {
        printf("error at %s:%d\n",__FILE__, __LINE__);
        return;
    }

}
// borrowed from https://stackoverflow.com/questions/21112725/cuda-fill-an-matrix-with-random-values-between-a-and-b
int iDivUp(int a, int b) { return ((a % b) != 0) ? (a / b + 1) : (a / b); }

__global__ void shiftDiagonal(double *A, double reg, int num_cols) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < num_cols) {
      A[num_cols*tid + tid] += reg;
    }
}


void least_squares (hipblasHandle_t handle, int* indptr, int* indic, double* data, int users,
                        int items, int factors,
                        double* x, double* y,
                        double reg, int istart, int iend)
{
    hipError_t err;
    hipblasStatus_t stat;
    hipsolverHandle_t solve_handle;
    hipsolverStatus_t solve_stat;
    const double alpha = 1;
    const double beta = 0;

    //double yt[factors][items];

    double* yt;

    err = hipMallocManaged(&yt, factors*items*sizeof(double));
    if (err != hipSuccess) {
      printf("%s\n", hipGetErrorString(err));
      hipFree(yt);
      return;
    }

    stat = hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, factors, items,
        &alpha, y, items, &beta, y, items, yt, factors);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("dgeam failed\n");
        hipFree(yt);
        return;
    }

    hipblasSideMode_t mode = HIPBLAS_SIDE_LEFT;

    for (int i =0; i < users; ++i) {
      if (i % 10000 == 0 || i == users-1) {
        printf("iteration %d\n", i);
      }
        //printf("starting for loop\n");
        // printf("users %d i %d\n", users, i);
        hipDeviceSynchronize();

        int rowStart = indptr[i];
        int rowEnd = indptr[i+1];

        if (rowStart == rowEnd) {
          printf("AW CRAP\n");
        }

        // printf("rowstart %d\n", rowStart);
        // printf("rowEnd %d\n", rowEnd);

        // printf("merp\n");
        int cols[rowEnd-rowStart];
        // printf("%d\n", indic[rowStart]);

        memcpy(cols, &indic[rowStart], (rowEnd-rowStart)*sizeof(int));
        // err = hipMemcpy(cols, ptrptr, rowEnd-rowStart, hipMemcpyDeviceToHost);
        // if (err != hipSuccess) {
        //   printf("first copy %s\n", hipGetErrorString(err));
        //   return;
        // }
        // printf("%d\n", cols[0]);
        // printf("done w first memcpy\n");
        double vals[rowEnd-rowStart];
        memcpy(vals, &data[rowStart], (rowEnd-rowStart)*sizeof(double));
        // err = hipMemcpy(vals, &data[rowStart], rowEnd-rowStart, hipMemcpyDeviceToHost);
        // if (err != hipSuccess) {
        //   printf("first copy %s\n", hipGetErrorString(err));
        //   return;
        // }
        // printf("done with memcpys\n");

        double diagVec[items];
        std::fill_n(diagVec, items, 0);
        // printf("items %d\n", items);
        //hipDeviceSynchronize();
        for(int iter = 0; iter < (rowEnd-rowStart); ++iter)
        {
            // printf("loooop\n");
            // printf("%d\n", cols[iter]);
            // printf("%f\n", vals[iter]);
            //printf("test\n");
            diagVec[cols[iter]] = vals[iter];
            //printf("success\n");
        }
        // printf("making device vec\n");
        double* diagVec_dev;
        hipDeviceSynchronize();
        err = hipMallocManaged(&diagVec_dev, items*sizeof(double));
        if (err != hipSuccess) {
          printf("%s\n", hipGetErrorString(err));
          hipFree(yt);
          hipFree(diagVec_dev);
          return;
        }
        err = hipMemcpy(diagVec_dev, diagVec, items*sizeof(double), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
          printf("%s\n", hipGetErrorString(err));
          hipFree(yt);
          hipFree(diagVec_dev);
          return;
        }
        // printf("making ansC\n");

        double* ansC; //[items][factors];

        err = hipMallocManaged(&ansC, items*factors*sizeof(double));
        if (err != hipSuccess) {
          printf("%s\n", hipGetErrorString(err));
          hipFree(yt);
          hipFree(diagVec_dev);
          hipFree(ansC);
          return;
        }
        hipDeviceSynchronize();
        // printf("dgmm\n");
        stat = hipblasDdgmm(handle, mode, items, factors, y, items, diagVec_dev, 1, ansC, items);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            printf ("ddgmm failed\n");
            hipFree(yt);
            hipFree(diagVec_dev);
            hipFree(ansC);
            return;
        }
        hipDeviceSynchronize();
        double* ytcy; //[factors][factors];

        err = hipMallocManaged(&ytcy, factors*factors*sizeof(double));
        if (err != hipSuccess) {
          printf("%s\n", hipGetErrorString(err));
          hipFree(yt);
          hipFree(diagVec_dev);
          hipFree(ansC);
          hipFree(ytcy);
          return;
        }

        // printf("gemm\n");
        stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, factors, factors,
                        items, &alpha, yt, factors, ansC, items, &beta, ytcy, factors);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            printf ("dgemm failed\n");
            // printf("%d\n", stat);
            hipFree(yt);
            hipFree(diagVec_dev);
            hipFree(ansC);
            hipFree(ytcy);
            return;
        }
        hipDeviceSynchronize();
        // printf("setting diagonal\n");
        // for (int k = 0; k < factors; ++k)
        // {
        //     printf("%d\n",k);
        //     printf("%f\n", ytcy[0]);
        //     ytcy[factors*k +k] += reg;
        // }
        // dim3 threads(32);
        // dim3 blocks(iDivUp(factors*factors, 32));
        // shiftDiagonal<<<blocks, threads>>>(ytcy, reg, factors);
        // printf("done w diagonal\n");
        //hipDeviceSynchronize();

        double* ytcu; //[factors];
        // printf("kjdfjkfdjkfd\n");
        err = hipMallocManaged(&ytcu, factors*sizeof(double));
        if (err != hipSuccess) {
          printf("%s\n", hipGetErrorString(err));
          hipFree(yt);
          hipFree(diagVec_dev);
          hipFree(ansC);
          hipFree(ytcy);
          hipFree(ytcu);
          return;
        }
        double host[factors];
        hipMemcpy(host, &ytcu[0], factors*sizeof(double), hipMemcpyDeviceToHost);
        for (int f = 0; f < factors; ++f) {
          printf("merp %f\n", host[f]);
        }
        hipDeviceSynchronize();
        // printf("gemv\n");
        stat = hipblasDgemv (handle, HIPBLAS_OP_N, factors, items,
                        &alpha, yt, factors, diagVec_dev, 1, &beta, ytcu, 1);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            printf ("dgemm failed\n");
            hipFree(yt);
            hipFree(diagVec_dev);
            hipFree(ansC);
            hipFree(ytcy);
            hipFree(ytcu);
            return;
        }
        hipDeviceSynchronize();
        // printf("solver fun times\n");
        solve_stat = hipsolverDnCreate(&solve_handle);
        if (HIPSOLVER_STATUS_SUCCESS != solve_stat) {
          printf("creating solver failed\n");
          hipFree(yt);
          hipFree(diagVec_dev);
          hipFree(ansC);
          hipFree(ytcy);
          hipFree(ytcu);
          return;
        }
        hipDeviceSynchronize();
        int Lwork;
        // printf("buffersize\n");
        solve_stat = hipsolverDnDpotrf_bufferSize(solve_handle, HIPBLAS_FILL_MODE_UPPER,
          factors, ytcy, factors, &Lwork);
        if (HIPSOLVER_STATUS_SUCCESS != solve_stat) {
          printf("get buffer size failed\n");
          hipFree(yt);
          hipFree(diagVec_dev);
          hipFree(ansC);
          hipFree(ytcy);
          hipFree(ytcu);
          return;
        }
        hipDeviceSynchronize();
        int* rfOut;
        err = hipMallocManaged(&rfOut, sizeof(int));
        if (err != hipSuccess) {
          printf("%s\n", hipGetErrorString(err));
          hipFree(yt);
          hipFree(diagVec_dev);
          hipFree(ansC);
          hipFree(ytcy);
          hipFree(ytcu);
          hipFree(rfOut);
          return;
        }
        double* workspace;
        err = hipMallocManaged(&workspace, Lwork*sizeof(double));
        if (err != hipSuccess) {
          printf("%s\n", hipGetErrorString(err));
          hipFree(yt);
          hipFree(diagVec_dev);
          hipFree(ansC);
          hipFree(ytcy);
          hipFree(ytcu);
          hipFree(rfOut);
          hipFree(workspace);
          return;
        }
        hipDeviceSynchronize();
        // printf("factorizing\n");
        solve_stat = hipsolverDnDpotrf(solve_handle, HIPBLAS_FILL_MODE_UPPER,
           factors, ytcy, factors, workspace, Lwork, rfOut);
        if (HIPSOLVER_STATUS_SUCCESS != solve_stat) {
          printf("factorization failed\n");
          hipFree(yt);
          hipFree(diagVec_dev);
          hipFree(ansC);
          hipFree(ytcy);
          hipFree(ytcu);
          hipFree(rfOut);
          hipFree(workspace);
          return;
        }
        //printf("%d\n", *rfOut);
        hipDeviceSynchronize();
        int* rsOut;
        err = hipMallocManaged(&rsOut, sizeof(int));
        if (err != hipSuccess) {
          printf("%s\n", hipGetErrorString(err));
          hipFree(yt);
          hipFree(diagVec_dev);
          hipFree(ansC);
          hipFree(ytcy);
          hipFree(ytcu);
          hipFree(rfOut);
          hipFree(workspace);
          hipFree(rsOut);
          return;
        }
        // printf("solving\n");
        solve_stat = hipsolverDnDpotrs(solve_handle, HIPBLAS_FILL_MODE_UPPER,
          factors, 1, ytcy, factors, ytcu, factors, rsOut);
        if (HIPSOLVER_STATUS_SUCCESS != solve_stat) {
          printf("solver failed\n");
          printf("err %d\n", solve_stat);
          // printf("HIPSOLVER_STATUS_SUCCESS = %d \n", HIPSOLVER_STATUS_SUCCESS);
          // printf("HIPSOLVER_STATUS_NOT_INITIALIZED = %d \n", HIPSOLVER_STATUS_NOT_INITIALIZED);
          // printf("HIPSOLVER_STATUS_INVALID_VALUE = %d \n", HIPSOLVER_STATUS_INVALID_VALUE);
          // printf("HIPSOLVER_STATUS_ARCH_MISMATCH = %d \n", HIPSOLVER_STATUS_ARCH_MISMATCH);
          // printf("HIPSOLVER_STATUS_INTERNAL_ERROR = %d \n", HIPSOLVER_STATUS_INTERNAL_ERROR);
          printf("out %d\n", *rsOut);
          hipFree(yt);
          hipFree(diagVec_dev);
          hipFree(ansC);
          hipFree(ytcy);
          hipFree(ytcu);
          hipFree(rfOut);
          hipFree(workspace);
          hipFree(rsOut);
          return;
        }
        hipDeviceSynchronize();
        // printf("doing memcpy\n");
        hipMemcpy(&x[i*factors], &ytcu[0], factors*sizeof(double), hipMemcpyDeviceToDevice);
        //double host[factors];
        hipMemcpy(host, &ytcu[0], factors*sizeof(double), hipMemcpyDeviceToHost);
        for (int f = 0; f < factors; ++f) {
          printf(" oh no %f\n", host[f]);
        }
        //x[i] = ytcu;
        hipDeviceSynchronize();
        // printf("freeing things\n");
        hipsolverDnDestroy(solve_handle);
        hipFree(diagVec_dev);
        hipFree(ansC);
        hipFree(ytcy);
        hipFree(ytcu);
        hipFree(rfOut);
        hipFree(workspace);
        hipFree(rsOut);
    }
    printf("done\n");
    hipFree(yt);
}
