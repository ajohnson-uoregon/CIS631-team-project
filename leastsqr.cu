#include "hip/hip_runtime.h"
#include "leastsqr.h"

void GPU_fill_rand(double *A, const int nr_rows_A, const int nr_cols_A)
{
    hiprandGenerator_t prng;
    hiprandStatus_t stat;

    stat = hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
    if (stat != HIPRAND_STATUS_SUCCESS) {
        printf("error at %s:%d\n",__FILE__, __LINE__);
        return;
    }
    stat = hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());
    if (stat != HIPRAND_STATUS_SUCCESS) {
        printf("error at %s:%d\n",__FILE__, __LINE__);
        return;
    }

    stat = hiprandGenerateUniformDouble(prng, A, nr_rows_A * nr_cols_A);
    if (stat != HIPRAND_STATUS_SUCCESS) {
        printf("error at %s:%d\n",__FILE__, __LINE__);
        return;
    }

}


void least_squares (hipblasHandle_t handle, int* indptr, int* indic, double* data, int users,
                        int items, int factors,
                        double* x, double* y,
                        double reg, int istart, int iend)
{
    hipError_t err;
    hipblasStatus_t stat;
    hipsolverHandle_t solve_handle;
    const double alpha = 1;
    const double beta = 0;

    //double yt[factors][items];

    double* yt;

    err = hipMallocManaged(&yt, factors*items*sizeof(double));
    if (err != hipSuccess) {
      printf("%s\n", hipGetErrorString(err));
      hipFree(yt);
      return;
    }

    stat = hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, factors, items,
        &alpha, y, items, &beta, y, items, yt, factors);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("dgeam failed\n");
        hipFree(yt);
        return;
    }

    hipblasSideMode_t mode = HIPBLAS_SIDE_LEFT;

    for (int i =0; i < users; ++i) {
        printf("starting for loop\n");
        printf("users %d i %d\n", users, i);

        int rowStartptr[1];
        err = hipMemcpy(rowStartptr, &indptr[i], 1, hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
          printf("first copy %s\n", hipGetErrorString(err));
          return;
        }
        int rowEndptr[1];
        err = hipMemcpy(rowEndptr, &indptr[i+1], 1, hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
          printf("second copy %s\n", hipGetErrorString(err));
          return;
        }

        int rowStart = rowStartptr[0];
        int rowEnd = rowEndptr[0];

        printf("rowstart %d\n", rowStart);
        printf("rowEnd %d\n", rowEnd);

        printf("merp\n");
        int cols[rowEnd-rowStart];
        err = hipMemcpy(cols, &indic[rowStart], rowEnd-rowStart, hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
          printf("first copy %s\n", hipGetErrorString(err));
          return;
        }
        printf("done w first memcpy\n");
        double vals[rowEnd-rowStart];
        err = hipMemcpy(vals, &data[rowStart], rowEnd-rowStart, hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
          printf("first copy %s\n", hipGetErrorString(err));
          return;
        }
        printf("done with memcpys\n");

        double diagVec[items];
        std::fill_n(diagVec, items, 0);

        for(int iter = 0; iter < (rowEnd-rowStart); ++iter)
        {
            diagVec[cols[iter]] = vals[iter];
        }
        printf("making device vec\n");
        double* diagVec_dev;

        err = hipMallocManaged(&diagVec_dev, items*sizeof(double));
        if (err != hipSuccess) {
          printf("%s\n", hipGetErrorString(err));
          hipFree(yt);
          hipFree(diagVec_dev);
          return;
        }
        err = hipMemcpy(diagVec_dev, diagVec, items, hipMemcpyHostToDevice);
        if (err != hipSuccess) {
          printf("%s\n", hipGetErrorString(err));
          hipFree(yt);
          hipFree(diagVec_dev);
          return;
        }
        printf("making ansC\n");

        double* ansC; //[items][factors];

        err = hipMallocManaged(&ansC, items*factors*sizeof(double));
        if (err != hipSuccess) {
          printf("%s\n", hipGetErrorString(err));
          hipFree(yt);
          hipFree(diagVec_dev);
          hipFree(ansC);
          return;
        }

        printf("dgmm\n");
        stat = hipblasDdgmm(handle, mode, items, factors, y, items, diagVec, 1, ansC, items);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            printf ("ddgmm failed\n");
            hipFree(yt);
            hipFree(diagVec_dev);
            hipFree(ansC);
            return;
        }
        double* ytcy; //[factors][factors];

        err = hipMallocManaged(&ytcy, factors*factors*sizeof(double));
        if (err != hipSuccess) {
          printf("%s\n", hipGetErrorString(err));
          hipFree(yt);
          hipFree(diagVec_dev);
          hipFree(ansC);
          hipFree(ytcy);
          return;
        }

        printf("gemm\n");
        stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, factors, factors,
                        items, &alpha, yt, factors, ansC, items, &beta, ytcy, factors);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            printf ("dgemm failed\n");
            hipFree(yt);
            hipFree(diagVec_dev);
            hipFree(ansC);
            hipFree(ytcy);
            return;
        }
        printf("setting diagonal\n");
        for (int k = 0; k < factors; ++k)
        {
            ytcy[factors*k +k] += reg;
        }

        double* ytcu; //[factors];
        err = hipMallocManaged(&ytcu, factors*sizeof(double));
        if (err != hipSuccess) {
          printf("%s\n", hipGetErrorString(err));
          hipFree(yt);
          hipFree(diagVec_dev);
          hipFree(ansC);
          hipFree(ytcy);
          hipFree(ytcu);
          return;
        }
        printf("gemv\n");
        stat = hipblasDgemv (handle, HIPBLAS_OP_N, factors, items,
                        &alpha, yt, factors, diagVec_dev, 1, &beta, ytcu, 1);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            printf ("dgemm failed\n");
            hipFree(yt);
            hipFree(diagVec_dev);
            hipFree(ansC);
            hipFree(ytcy);
            hipFree(ytcu);
            return;
        }
        printf("solver fun times\n");
        int Lwork;
        hipsolverDnDgetrf_bufferSize(solve_handle, factors, factors, ytcy, factors, &Lwork);
        int* rfOut;
        double workspace[Lwork];
        hipsolverDnDgetrf(solve_handle, factors, factors, ytcy, factors, workspace, NULL, rfOut);
        int* rsOut;
        hipsolverDnDgetrs(solve_handle, HIPBLAS_OP_N, factors, 1, ytcy, factors, NULL, ytcu, factors, rsOut);
        memcpy(&x[(i*factors)], ytcu, factors);
        //x[i] = ytcu;
        hipFree(diagVec_dev);
        hipFree(ansC);
        hipFree(ytcy);
        hipFree(ytcu);
    }
    printf("done\n");
    hipFree(yt);
}
