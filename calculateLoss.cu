#ifndef calculate_loss_CU
#define calculate_loss_CU

#include <iostream>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "hipsparse.h"
#include <cmath>
#include <vector>
#include <fstream>
#include <iomanip>
#include <cstring>

double calculate_loss(int* indptr, int* indices, double* data, double** X, double** Y, double reg, 
    int users, int items, int factors, int nnz) {
  
      int loss = 0;
      int total_confidence = 0;
      int item_norm = 0;
      int user_norm = 0;
      hipblasStatus_t err;
      hipblasHandle_t handle;
  
      double** YtY;
      const double alpha = 1;
      const double beta = 0;
  
      // do transpose
      err = hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, items, factors,
                &alpha, Y[0], items, &beta, Y[0], items, YtY[0], factors);
      
      for (int u = 0; u < users; ++u) {
          double temp = 1.0;
  
          double* r;
          double* Xu = X[u];
          
  
          err = hipblasDgemv(handle, HIPBLAS_OP_N, items, factors,
                    &alpha, Y[0], items, Xu, 1, &beta, r, 1);
          
          int rowStart = indptr[u];
          int rowEnd = indptr[u+1];
  
          int cols[rowEnd-rowStart];
          memcpy(cols, &indices[rowStart], rowEnd-rowStart);
          //int* cols = Cui[1][rowStart:rowEnd];
          
          double vals[rowEnd-rowStart];
          memcpy(vals, &data[rowStart], rowEnd-rowStart);
          //double* vals = Cui[2][rowStart:rowEnd];
          for (int index = 0; index < rowEnd-rowStart; ++index) {
              int i = cols[index];
              double confidence = vals[index];
  
              double* Yi = Y[i];
  
              double d;
              err = hipblasDdot(handle, factors, Yi, 1, Xu, 1, &d);
              temp = (confidence - 1)*d - (2*confidence);
              
              err = hipblasDaxpy(handle, factors, &temp, Yi, 1, r, 1);
              total_confidence += confidence;
              loss += confidence;
          }
  
          double other_temp;
  
          err = hipblasDdot(handle, factors, r, 1, Xu, 1, &other_temp);
          loss += other_temp;
          
          err = hipblasDdot(handle, factors, Xu, 1, Xu, 1, &other_temp);
          user_norm += other_temp;
      }
  
      for (int i = 0; i < items; ++i) {
          double* Yi = Y[i];
  
          double other_temp;
   
          err = hipblasDdot(handle, factors, Yi, 1, Yi, 1, &other_temp);
          item_norm += other_temp;
      }
  
      loss += reg * (item_norm + user_norm);
  
      return loss / (total_confidence + users * items - nnz);
  }

#endif
