#include "calculateLoss.h"

double calculate_loss(hipblasHandle_t handle, int* indptr, int* indices, double* data, double* X, double* Y, double reg,
    int users, int items, int factors, int nnz) {

      int loss = 0;
      int total_confidence = 0;
      int item_norm = 0;
      int user_norm = 0;
      hipblasStatus_t stat;
      hipError_t err;

      // malloc this
      double* YtY;
      err = hipMallocManaged(&YtY, factors*factors*sizeof(double));
      if (err != hipSuccess) {
        printf("%s\n", hipGetErrorString(err));
        hipFree(YtY);
        return -1;
      }
      const double alpha = 1;
      const double beta = 0;

      // do transpose
      stat = hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, factors, factors, items,
                &alpha, Y, items, Y, items, &beta, YtY, factors);
      if (stat != HIPBLAS_STATUS_SUCCESS) {
          printf ("dgemm failed\n");
          hipFree(YtY);
          return -1;
      }

      for (int u = 0; u < users; ++u) {
        if (u % 10000 == 0) {
          printf("loss iter %d\n", u);
        }
          hipDeviceSynchronize();
          double temp = 1.0;

          double* r;
          err = hipMallocManaged(&r, items*sizeof(double));
          if (err != hipSuccess) {
            printf("%s\n", hipGetErrorString(err));
            hipFree(YtY);
            hipFree(r);
            return -1;
          }
          double* Xu = &X[u*factors];

          stat = hipblasDgemv(handle, HIPBLAS_OP_N, items, factors,
                    &alpha, Y, items, Xu, 1, &beta, r, 1);
          if (stat != HIPBLAS_STATUS_SUCCESS) {
              printf ("dgemv failed\n");
              hipFree(YtY);
              hipFree(r);
              return -1;
          }
          hipDeviceSynchronize();

          int rowStart = indptr[u];
          int rowEnd = indptr[u+1];

          int cols[rowEnd-rowStart];
          memcpy(cols, &indices[rowStart], (rowEnd-rowStart)*sizeof(int));
          //int* cols = Cui[1][rowStart:rowEnd];

          double vals[rowEnd-rowStart];
          memcpy(vals, &data[rowStart], (rowEnd-rowStart)*sizeof(double));
          //double* vals = Cui[2][rowStart:rowEnd];
          for (int index = 0; index < rowEnd-rowStart; ++index) {
              int i = cols[index];
              double confidence = vals[index];

              double* Yi = &Y[i*factors];
              hipDeviceSynchronize();
              double d;
              stat = hipblasDdot(handle, factors, Yi, 1, Xu, 1, &d);
              if (stat != HIPBLAS_STATUS_SUCCESS) {
                  printf ("ddot 1 failed\n");
                  hipFree(YtY);
                  hipFree(r);
                  return -1;
              }
              temp = (confidence - 1)*d - (2*confidence);
              hipDeviceSynchronize();
              stat = hipblasDaxpy(handle, factors, &temp, Yi, 1, r, 1);
              if (stat != HIPBLAS_STATUS_SUCCESS) {
                  printf ("daxpy failed\n");
                  hipFree(YtY);
                  hipFree(r);
                  return -1;
              }
              total_confidence += confidence;
              loss += confidence;
          }

          double other_temp;
          hipDeviceSynchronize();
          stat = hipblasDdot(handle, factors, r, 1, Xu, 1, &other_temp);
          if (stat != HIPBLAS_STATUS_SUCCESS) {
              printf ("ddot 2 failed\n");
              hipFree(YtY);
              hipFree(r);
              return -1;
          }
          loss += other_temp;
          hipDeviceSynchronize();
          stat = hipblasDdot(handle, factors, Xu, 1, Xu, 1, &other_temp);
          if (stat != HIPBLAS_STATUS_SUCCESS) {
              printf ("ddot 3 failed\n");
              hipFree(YtY);
              hipFree(r);
              return -1;
          }
          user_norm += other_temp;

          hipFree(r);
      }

      for (int i = 0; i < items; ++i) {
          hipDeviceSynchronize();
          double* Yi = &Y[i*factors];

          double other_temp;

          stat = hipblasDdot(handle, factors, Yi, 1, Yi, 1, &other_temp);
          if (stat != HIPBLAS_STATUS_SUCCESS) {
              printf ("ddot 4 failed\n");
              hipFree(YtY);
              return -1;
          }
          item_norm += other_temp;
      }

      loss += reg * (item_norm + user_norm);

      hipFree(YtY);

      return loss / ((double) (total_confidence + users * items - nnz));
  }
