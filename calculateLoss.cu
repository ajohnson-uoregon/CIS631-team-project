#include "calculateLoss.h"

double calculate_loss(hipblasHandle_t handle, int* indptr, int* indices, double* data, double* X, double* Y, double reg,
    int users, int items, int factors, int nnz) {

      int loss = 0;
      int total_confidence = 0;
      int item_norm = 0;
      int user_norm = 0;
      hipblasStatus_t err;

      // malloc this
      double** YtY;
      const double alpha = 1;
      const double beta = 0;

      // do transpose
      err = hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, items, factors,
                &alpha, Y, items, &beta, Y, items, YtY[0], factors);


      for (int u = 0; u < users; ++u) {
          hipDeviceSynchronize();
          double temp = 1.0;

          double* r;
          double* Xu = &X[u*factors];


          err = hipblasDgemv(handle, HIPBLAS_OP_N, items, factors,
                    &alpha, Y, items, Xu, 1, &beta, r, 1);
          hipDeviceSynchronize();

          int rowStart = indptr[u];
          int rowEnd = indptr[u+1];

          int cols[rowEnd-rowStart];
          memcpy(cols, &indices[rowStart], (rowEnd-rowStart)*sizeof(int));
          //int* cols = Cui[1][rowStart:rowEnd];

          double vals[rowEnd-rowStart];
          memcpy(vals, &data[rowStart], (rowEnd-rowStart)*sizeof(double));
          //double* vals = Cui[2][rowStart:rowEnd];
          for (int index = 0; index < rowEnd-rowStart; ++index) {
              int i = cols[index];
              double confidence = vals[index];

              double* Yi = &Y[i*factors];
              hipDeviceSynchronize();
              double d;
              err = hipblasDdot(handle, factors, Yi, 1, Xu, 1, &d);
              temp = (confidence - 1)*d - (2*confidence);
              hipDeviceSynchronize();
              err = hipblasDaxpy(handle, factors, &temp, Yi, 1, r, 1);
              total_confidence += confidence;
              loss += confidence;
          }

          double other_temp;
          hipDeviceSynchronize();
          err = hipblasDdot(handle, factors, r, 1, Xu, 1, &other_temp);
          loss += other_temp;
          hipDeviceSynchronize();
          err = hipblasDdot(handle, factors, Xu, 1, Xu, 1, &other_temp);
          user_norm += other_temp;
      }

      for (int i = 0; i < items; ++i) {
          hipDeviceSynchronize();
          double* Yi = &Y[i*factors];

          double other_temp;

          err = hipblasDdot(handle, factors, Yi, 1, Yi, 1, &other_temp);
          item_norm += other_temp;
      }

      loss += reg * (item_norm + user_norm);

      return loss / (total_confidence + users * items - nnz);
  }
